#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hip/hip_complex.h>        // hipDoubleComplex
#include <cutensornet.h>

#include <vector>

#include "./cnot.h"
#include "./h.h"

int main(void) {

  const int32_t n_qubits = 1;

  const std::vector<int64_t> qubits (n_qubits, 2);

  hipSetDevice(0);
  cutensornetHandle_t handle;
  cutensornetCreate(&handle);

  void *h_gate_for_device{nullptr};

  hipMalloc(&h_gate_for_device, 4*sizeof(hipDoubleComplex));
  hipMemcpy(h_gate_for_device, H, 4*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  


  return EXIT_SUCCESS;
}
