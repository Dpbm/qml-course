#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <custatevec.h>
#include <iostream>
#include <cmath>

#include "./x.h"
#include "./h.h"

using std::cout;
using std::endl;

int main(void)
{
  hipSetDevice(0);

  int n_devices;
  hipError_t cuda_error = hipGetDeviceCount(&n_devices);

  if (cuda_error != hipSuccess)
  {
    cout << "hipGetDeviceCount Error: "
         << hipGetErrorString(cuda_error) << endl;
  }
  else
  {
    cout << "Total CUDA Devices: " << n_devices << endl;
  }
	
  custatevecHandle_t handle;
  custatevecCreate(&handle);

  const int n_qubits = 2;
  const int state_size = std::pow(2, n_qubits);

  hipDoubleComplex *state = new hipDoubleComplex[state_size];

  hipDoubleComplex *statevector;
  hipMalloc((void **)&statevector, state_size * sizeof(hipDoubleComplex));

  hipMemcpy(
      statevector,
      state,
      state_size * sizeof(hipDoubleComplex),
      hipMemcpyHostToDevice);


  custatevecInitializeStateVector(handle, statevector, HIP_C_64F, n_qubits, CUSTATEVEC_STATE_VECTOR_TYPE_ZERO);
  
  int hadamard_target[] = {0};
  custatevecApplyMatrix(
      handle,
      statevector,
      HIP_C_64F,
      1,//n_qubits
      H, //hadamard gate
      HIP_C_64F,
      CUSTATEVEC_MATRIX_LAYOUT_COL,
      0, //no adjoint
      hadamard_target, //target
      1, //n_targets
      nullptr, // controls
      nullptr, // control bit-string
      0,       // n controls
      CUSTATEVEC_COMPUTE_64F,
      nullptr,
      0);

  int cnot_target[] = {1};
  int cnot_control[] = {0};
  custatevecApplyMatrix(
      handle,
      statevector,
      HIP_C_64F,
      n_qubits,//n_qubits
      X, //cnot(x controlled) gate
      HIP_C_64F,
      CUSTATEVEC_MATRIX_LAYOUT_COL,
      0, //no adjoint
      cnot_target, //target
      1, //n_targets
      cnot_control, // controls
      nullptr, // control bit-string
      1,       // n controls
      CUSTATEVEC_COMPUTE_64F,
      nullptr,
      0);

  hipMemcpy(
      state,
      statevector,
      state_size * sizeof(hipDoubleComplex),
      hipMemcpyDeviceToHost);

  for (int i = 0; i < state_size; i++)
  {
    hipDoubleComplex c = state[i];
    cout << (double)c.x << endl;
  }

  custatevecDestroy(handle);
  hipFree(statevector);
  delete state;

  return 0;
}
