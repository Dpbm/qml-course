#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_complex.h>
#include <custatevec.h>
#include <iostream>
#include <cmath>

#include "./h.h"

using std::cout;
using std::endl;

int main(void)
{

  hipSetDevice(0);

  int n_devices;
  hipError_t cuda_error = hipGetDeviceCount(&n_devices);

  if (cuda_error != hipSuccess)
  {
    cout << "hipGetDeviceCount Error: "
         << hipGetErrorString(cuda_error) << endl;
  }
  else
  {
    cout << "Total CUDA Devices: " << n_devices << endl;
  }

  custatevecHandle_t handle;
  custatevecCreate(&handle);

  const int n_qubits = 1;
  const int n_targets = 1;
  const int n_controls = 0;
  const int adjoint = 0;
  const int state_size = std::pow(2, n_qubits);

  hipDoubleComplex *state = new hipDoubleComplex[state_size];

  hipDoubleComplex *statevector;
  hipMalloc((void **)&statevector, state_size * sizeof(hipDoubleComplex));

  hipDoubleComplex *hadamard;
  hipMalloc((void **)&hadamard, 4 * sizeof(hipDoubleComplex));

  hipMemcpy(
      statevector,
      state,
      state_size * sizeof(hipDoubleComplex),
      hipMemcpyHostToDevice);

  hipMemcpy(
      hadamard,
      H,
      4 * sizeof(hipDoubleComplex),
      hipMemcpyHostToDevice);

  cout << "H Matrix" << endl;
  for (int i = 0; i < 4; i++)
  {
    hipDoubleComplex c = H[i];
    cout << (double)c.x << endl;
  }

  custatevecInitializeStateVector(handle, statevector, HIP_C_64F, n_qubits, CUSTATEVEC_STATE_VECTOR_TYPE_ZERO);

  int targets[] = {0};

  custatevecApplyMatrix(
      handle,
      statevector,
      HIP_C_64F,
      n_qubits,
      hadamard,
      HIP_C_64F,
      CUSTATEVEC_MATRIX_LAYOUT_COL,
      adjoint,
      targets,
      n_targets,
      nullptr, // controls
      nullptr, // control bit-string
      0,       // n controls
      CUSTATEVEC_COMPUTE_64F,
      nullptr,
      0);

  hipMemcpy(
      state,
      statevector,
      state_size * sizeof(hipDoubleComplex),
      hipMemcpyDeviceToHost);

  for (int i = 0; i < state_size; i++)
  {
    hipDoubleComplex c = state[i];
    cout << (double)c.x << endl;
  }

  custatevecDestroy(handle);
  hipFree(statevector);
  hipFree(hadamard);
  delete state;

  return 0;
}
